

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8
#define numThread 2 // 2 threads in a block
#define numBlock 4  // 4 blocks

/*
 * 1.
 *  The 'kernel' function that will be executed on the GPU device hardware.
 */
__global__ void add( int *a, int *b, int *c ) {

    // the initial index that this thread will work on
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];       // The actual computation done by the thread
        tid += blockDim.x;       // Increment this thread's index by the number of threads per block:
                                 // in this small case, each thread would then have a tid > N
    }
}
/*
 * The main program that directs the execution of vector add on the GPU
 */
int main( void ) {
    int *a, *b, *c;               // The arrays on the host CPU machine
    int *dev_a, *dev_b, *dev_c;   // The arrays for the GPU device

    // 2.a allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );

    // 2.b. fill the arrays 'a' and 'b' on the CPU with dummy values
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i;
    }

    // 2.c. allocate the memory on the GPU
     hipMalloc( (void**)&dev_a, N * sizeof(int) );
     hipMalloc( (void**)&dev_b, N * sizeof(int) );
     hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // 2.d. copy the arrays 'a' and 'b' to the GPU
     hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice );
     hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice );

    // 3. Execute the vector addition 'kernel function' on th GPU device,
    // declaring how many blocks and how many threads per block to use.
    add<<<numBlock,numThread>>>( dev_a, dev_b, dev_c );

    // 4. copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost );

    // verify that the GPU did the work we requested
    bool success = true;
    int total=0;
    printf("Checking %d values in the array.\n", N);
    for (int i=0; i<N; i++) {
        if ( c[i] = (a[i] + b[i])) {
            printf( "Addition: %d + %d = %d\n", a[i], b[i], c[i] );
            success = false;
        }
        total += 1;
    }
    if (success)  printf( "We did it, %d values correct!\n", total );

    // free the memory we allocated on the CPU
    free( a );
    free( b );
    free( c );

    // free the memory we allocated on the GPU
     hipFree( dev_a );
     hipFree( dev_b );
     hipFree( dev_c );

    return 0;
}
